#include "hip/hip_runtime.h"
#include <stdio.h>
// Macro for converting subscripts to linear index:
#define INDEX_VOL_TIME(i, t) i*${T} +t
#define INDEX_JAC_TIME(i, t, p) i*${T} *${D} +t*${D} +p
#define INDEX_PARAM(i, p) i*${D} +p
#define INDEX_MASK(x, y, z) x*${W} *${L} +y*${L} +z

/*******************************************************************************************************************************
                          MODEL FUNCTIONS DECLARATION (see the end of this file for the body of the functions)
*******************************************************************************************************************************/
__device__ void bicomp_3expIF(unsigned int idx, float *aux_par, float *inputfuns, float *IF, float *times, float *func, float *jac, float *dk);
__device__ void bicomp_2expIF_noDecay(unsigned int idx, float *aux_par, float *inputfuns, float *IF, float *times, float *func, float *jac, float *mask);

/*******************************************************************************************************************************
                                                       PET
*******************************************************************************************************************************/

// BICOMPARTMENT MODEL WITH IF MODELED AS SUM OF 3 EXP (like in Feng model #2)
__global__ void bicompartment_3expIF_4k(float *aux_par, float *inputfun, float *IF, float *time, float *func, float *jac, float *dk, float *mask)
{
	// Obtain the linear index corresponding to the current thread:
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	//*** Uncomment the lines below to use shared memory
	__shared__ float times[${T}];
	__shared__ float inputfuns[7];
	if (threadIdx.x < ${T}) {
		times[threadIdx.x] = time[threadIdx.x];
		if (threadIdx.x < 7)
			inputfuns[threadIdx.x] = inputfun[threadIdx.x];
	}
	__syncthreads();
	//*** Comment the lines above and uncomment below to disable shared memory
	//float *times = time;
	//float *inputfuns = inputfun;
	//***

	bicomp_3expIF(idx, aux_par, inputfuns, IF, times, func, jac, dk);
	//__syncthreads();
}

// BICOMPARTMENT MODEL WITH IF MODELED AS SUM OF 3 EXP (like in Feng model #2)
__global__ void bicompartment_3expIF_3k(float *aux_par, float *inputfun, float *IF, float *time, float *func, float *jac, float *dk, float *mask)
{
	// Obtain the linear index corresponding to the current thread:
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	//*** Uncomment the lines below to use shared memory
	__shared__ float times[${T}];
	__shared__ float inputfuns[7];
	if (threadIdx.x < ${T}) {
		times[threadIdx.x] = time[threadIdx.x];
		if (threadIdx.x < 7)
			inputfuns[threadIdx.x] = inputfun[threadIdx.x];
	}
	__syncthreads();
	//*** Comment the lines above and uncomment below to disable shared memory
	//float *times = time;
	//float *inputfuns = inputfun;
	//***

	bicomp_3expIF(idx, aux_par, inputfuns, IF, times, func, jac, dk);
	//__syncthreads();

	// deactivate the jacobian for the 4th kinetic constant we don't want to update
	for (uint tt=0; tt<${T}; ++tt) {
		jac[INDEX_JAC_TIME(idx,tt,4)] = 0;
	}
	//__syncthreads();
}

// MONOOMPARTMENT MODEL WITH IF MODELED AS SUM OF 2 EXP (like in Feng model #4)
__global__ void monocompartment_3expIF(float *aux_par, float *inputfun, float *IF, float *time, float *func, float *jac, float *dk, float *mask)
{
	// Obtain the linear index corresponding to the current thread:
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	//*** Uncomment the lines below to use shared memory
	__shared__ float times[${T}];
	__shared__ float inputfuns[5];
	if (threadIdx.x < ${T}) {
		times[threadIdx.x] = time[threadIdx.x];
		if (threadIdx.x < 5)
			inputfuns[threadIdx.x] = inputfun[threadIdx.x];
	}
	__syncthreads();
	//*** Comment the lines above and uncomment below to disable shared memory
	//float *times = time;
	//float *inputfuns = inputfun;
	//***

	bicomp_3expIF(idx, aux_par, inputfuns, IF, times, func, jac, dk);
	//__syncthreads();

	// deactivate the jacobian for the 4th kinetic constant we don't want to update
	for (uint tt=0; tt<${T}; ++tt) {
		jac[INDEX_JAC_TIME(idx,tt,3)] = 0;
		jac[INDEX_JAC_TIME(idx,tt,4)] = 0;
	}
	//__syncthreads();
}

/*******************************************************************************************************************************
                                                      DCE-MRI
*******************************************************************************************************************************/

// BICOMPARTMENT MODEL WITH IF MODELED AS SUM OF 2 EXP (like in Feng model #4)
__global__ void bicompartment_2expIF_4k(float *aux_par, float *inputfun, float *IF, float *time, float *func, float *jac, float *dk, float *mask)
{
	// Obtain the linear index corresponding to the current thread:
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	//*** Uncomment the lines below to use shared memory
	__shared__ float times[${T}];
	__shared__ float inputfuns[5];
	if (threadIdx.x < ${T}) {
		times[threadIdx.x] = time[threadIdx.x];
		if (threadIdx.x < 5)
			inputfuns[threadIdx.x] = inputfun[threadIdx.x];
	}
	__syncthreads();
	//*** Comment the lines above and uncomment below to disable shared memory
	//float *times = time;
	//float *inputfuns = inputfun;
	//***

	bicomp_2expIF_noDecay(idx, aux_par, inputfuns, IF, times, func, jac, mask);
	//__syncthreads();
}

// BICOMPARTMENT MODEL WITH IF MODELED AS SUM OF 2 EXP (like in Feng model #4)
__global__ void bicompartment_2expIF_3k(float *aux_par, float *inputfun, float *IF, float *time, float *func, float *jac, float *dk, float *mask)
{
	// Obtain the linear index corresponding to the current thread:
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	//*** Uncomment the lines below to use shared memory
	__shared__ float times[${T}];
	__shared__ float inputfuns[5];
	if (threadIdx.x < ${T}) {
		times[threadIdx.x] = time[threadIdx.x];
		if (threadIdx.x < 5)
			inputfuns[threadIdx.x] = inputfun[threadIdx.x];
	}
	__syncthreads();
	//*** Comment the lines above and uncomment below to disable shared memory
	//float *times = time;
	//float *inputfuns = inputfun;
	//***

	bicomp_2expIF_noDecay(idx, aux_par, inputfuns, IF, times, func, jac, mask);
	//__syncthreads();

	// deactivate the jacobian for the 4th kinetic constant we don't want to update
	for (uint tt=0; tt<${T}; ++tt) {
		jac[INDEX_JAC_TIME(idx,tt,4)] = 0;
	}
	//__syncthreads();
}

// MONOOMPARTMENT MODEL WITH IF MODELED AS SUM OF 2 EXP (like in Feng model #4)
__global__ void monocompartment_2expIF(float *aux_par, float *inputfun, float *IF, float *time, float *func, float *jac, float *dk, float *mask)
{
	// Obtain the linear index corresponding to the current thread:
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	//*** Uncomment the lines below to use shared memory
	__shared__ float times[${T}];
	__shared__ float inputfuns[5];
	if (threadIdx.x < ${T}) {
		times[threadIdx.x] = time[threadIdx.x];
		if (threadIdx.x < 5)
			inputfuns[threadIdx.x] = inputfun[threadIdx.x];
	}
	__syncthreads();
	//*** Comment the lines above and uncomment below to disable shared memory
	//float *times = time;
	//float *inputfuns = inputfun;
	//***

	bicomp_2expIF_noDecay(idx, aux_par, inputfuns, IF, times, func, jac, mask);
	//__syncthreads();

	// deactivate the jacobian for the 4th kinetic constant we don't want to update
	for (uint tt=0; tt<${T}; ++tt) {
		jac[INDEX_JAC_TIME(idx,tt,3)] = 0;
		jac[INDEX_JAC_TIME(idx,tt,4)] = 0;
	}
	//__syncthreads();
}


/*******************************************************************************************************************************
                                                 COMPARTMENTAL MODELS IMPLEMENTATION
*******************************************************************************************************************************/

// ANALYTIC FORMULATION OF A BICOMPARTMENT MODEL WITH IF MODELED AS SUM OF 3 EXP (like in Feng model #2)
__device__ void bicomp_3expIF(unsigned int idx, float *aux_par, float *inputfuns, float *IF, float *times, float *func, float *jac, float *dk)
{
	//float s;
	//float d;
	float delta0;
	float delta;
	float p[4];
	float Ahat[3];
	float Abar[3];
	float sum[${T}];
	float TAC[${T}];
	float Jb[${T}];
	float Jl[${T}];
	//__syncthreads();

	// Compute output of bicompartmental model and Jacobian using analytical expression.
	if (idx < ${N}) {
		/* Auxiliary parameters

		   s = k[INDEX_PARAM(idx,2)] + k[INDEX_PARAM(idx,3)] + k[INDEX_PARAM(idx,4)];
		   d = abs(sqrt(s*s - 4*k[INDEX_PARAM(idx,2)]*k[INDEX_PARAM(idx,4)]));
		   p[1] = (s + d) / 2;   //L1
		   p[3] = (s - d) / 2;   //L2
		   p[0] = (k[INDEX_PARAM(idx,1)] * ( p[1] - k[INDEX_PARAM(idx,3)] - k[INDEX_PARAM(idx,4)])) / d;  //B1
		   p[2] = (k[INDEX_PARAM(idx,1)] * (-p[3] + k[INDEX_PARAM(idx,3)] + k[INDEX_PARAM(idx,4)])) / d;  //B2 */
		p[1] = aux_par[INDEX_PARAM(idx,2)];
		p[3] = aux_par[INDEX_PARAM(idx,4)];
		p[0] = aux_par[INDEX_PARAM(idx,1)];
		p[2] = aux_par[INDEX_PARAM(idx,3)];
		Abar[0] = -inputfuns[2]-inputfuns[3];
		Abar[1] =  inputfuns[2];
		Abar[2] =  inputfuns[3];

		for (uint tt=0; tt<${T}; ++tt) { // reset the values of TAC and JAC for current voxel/thread
			func[INDEX_VOL_TIME(idx,tt)] = 0;
			jac[INDEX_JAC_TIME(idx,tt,0)] = 0;
			jac[INDEX_JAC_TIME(idx,tt,1)] = 0;
			jac[INDEX_JAC_TIME(idx,tt,2)] = 0;
			jac[INDEX_JAC_TIME(idx,tt,3)] = 0;
			jac[INDEX_JAC_TIME(idx,tt,4)] = 0;
			TAC[tt] = 0;
		}
		//__syncthreads();

		for (uint ii=0; ii<=2; ii+=2) { //i = 1:2:4 % 2 compartiments
			delta0  = p[ii+1] + inputfuns[4];
			Ahat[0] = -inputfuns[2]-inputfuns[3]-(inputfuns[1]/delta0);
			Ahat[1] = inputfuns[2];
			Ahat[2] = inputfuns[3];

			for (uint tt=0; tt<${T}; ++tt) { // reset temporary variables for i-th compartment
				sum[tt]=0;
				Jb[tt] =0;
				Jl[tt] =0;
			}
			for (uint jj=0; jj<3; ++jj) {
				delta  = p[ii+1]+inputfuns[4+jj];

				for (uint tt=0; tt<${T}; ++tt) {
					if (times[tt]>=inputfuns[0]) {
						sum[tt] += Ahat[jj] * (1.0f / delta) * ( exp(inputfuns[4+jj]*(times[tt]-inputfuns[0]))-exp(-p[ii+1]*(times[tt]-inputfuns[0])) );
						Jb[tt]  += Ahat[jj] * (1.0f / delta) * ( exp(inputfuns[4+jj]*(times[tt]-inputfuns[0]))-exp(-p[ii+1]*(times[tt]-inputfuns[0])) );
						Jl[tt]  += Abar[jj] * (1.0f / (delta*delta)) * ( exp(-p[ii+1]*(times[tt]-inputfuns[0]))-exp(inputfuns[4+jj]*(times[tt]-inputfuns[0]))) + Abar[jj] * (1.0f / delta)*(times[tt]-inputfuns[0]) * exp(-p[ii+1]*(times[tt]-inputfuns[0]));
					}
				}
			}

			for (uint tt=0; tt<${T}; ++tt) {
				if (times[tt]>=inputfuns[0]) {
					TAC[tt] += p[ii] * (sum[tt] + ((inputfuns[1]*(times[tt]-inputfuns[0]))/delta0) *exp(inputfuns[4]*(times[tt]-inputfuns[0])));
					jac[INDEX_JAC_TIME(idx,tt,ii+1)] = (1-aux_par[INDEX_PARAM(idx,0)]) * (Jb[tt] + ((inputfuns[1]*(times[tt]-inputfuns[0]))/delta0) *exp(inputfuns[4]*(times[tt]-inputfuns[0])));
					jac[INDEX_JAC_TIME(idx,tt,ii+2)] = (1-aux_par[INDEX_PARAM(idx,0)]) * (p[ii] * (Jl[tt] + ( exp(-p[ii+1]*(times[tt]-inputfuns[0]))-exp(inputfuns[4]*(times[tt]-inputfuns[0]))) * (inputfuns[1] *(times[tt]-inputfuns[0]) * (1.0f / (delta0*delta0)) + 2*inputfuns[1] * (1.0f / (delta0*delta0*delta0))) ));
				}
			}

		}
		//__syncthreads();
		for (uint tt=0; tt<${T}; ++tt) {
			TAC[tt]  *= exp(-dk[0]*times[tt]);
			jac[INDEX_JAC_TIME(idx,tt,0)] = IF[tt] - TAC[tt];
			TAC[tt]  = ((1-aux_par[INDEX_PARAM(idx,0)]) * TAC[tt]) + (aux_par[INDEX_PARAM(idx,0)] * IF[tt]);
			if (TAC[tt] < 0.0) {
				TAC[tt] = 1e-16;
			}
			func[INDEX_VOL_TIME(idx,tt)] += TAC[tt];
		}
		//__syncthreads();
	}
}

// ANALYTIC FORMULATION OF A BICOMPARTMENT MODEL WITH IF MODELED AS SUM OF 2 EXP (like in Feng model #4) -- NO DECAY CORRECTION FOR DCE-MRI
__device__ void bicomp_2expIF_noDecay(unsigned int idx, float *aux_par, float *inputfuns, float *IF, float *times, float *func, float *jac, float *mask)
{
	float delta0;
	float delta;
	float p[4];
	float Ahat[2];
	float Abar[2];
	float sum[${T}];
	float TAC[${T}];
	float Jb[${T}];
	float Jl[${T}];

	unsigned int x = idx/(${W} *${L});
	unsigned int y = (idx%(${W} *${L}))/${L};
	unsigned int z = (idx%(${W} *${L}))%${L};
	// __syncthreads();

	// Compute output of bicompartmental model and Jacobian using analytical expression.
	if (idx < ${N}) {
		/* Auxiliary parameters

		   p[1] = k[INDEX_PARAM(idx,2)] + k[INDEX_PARAM(idx,3)];   //L1
		   p[3] = 0;   //L2
		   p[0] = k[INDEX_PARAM(idx,1)] *  k[INDEX_PARAM(idx,2)] / p[1];  //B1
		   p[2] = k[INDEX_PARAM(idx,1)] *  k[INDEX_PARAM(idx,3)] / p[1];  //B2 */
		p[1] = aux_par[INDEX_PARAM(idx,2)];
		p[3] = aux_par[INDEX_PARAM(idx,4)];
		p[0] = aux_par[INDEX_PARAM(idx,1)];
		p[2] = aux_par[INDEX_PARAM(idx,3)];
		Abar[0] = -inputfuns[2];
		Abar[1] =  inputfuns[2];

		for (uint tt=0; tt<${T}; ++tt) { // reset the values of TAC and JAC for current voxel/thread
			func[INDEX_VOL_TIME(idx,tt)] = 0;
			jac[INDEX_JAC_TIME(idx,tt,0)] = 0;
			jac[INDEX_JAC_TIME(idx,tt,1)] = 0;
			jac[INDEX_JAC_TIME(idx,tt,2)] = 0;
			jac[INDEX_JAC_TIME(idx,tt,3)] = 0;
			jac[INDEX_JAC_TIME(idx,tt,4)] = 0;
			TAC[tt] = 0;
		}
		//__syncthreads();
		if ((idx*mask[INDEX_MASK(x, y, z)]!= 0) || (idx == 0 && mask[INDEX_MASK(x, y, z)]!= 0)) {
			for (uint ii=0; ii<=2; ii+=2) { //i = 1:2:4 % 2 compartiments
				delta0  = 1.0f / (p[ii+1] + inputfuns[3]);
				Ahat[0] = -inputfuns[2]-(inputfuns[1]/delta0);
				Ahat[1] = inputfuns[2];

				for (uint tt=0; tt<${T}; ++tt) { // reset temporary variables for i-th compartment
					sum[tt]=0;
					Jb[tt] =0;
					Jl[tt] =0;
				}
				for (uint jj=0; jj<2; ++jj) {
					delta  = 1.0f / (p[ii+1]+inputfuns[3+jj]);

					for (uint tt=0; tt<${T}; ++tt) {
						if (times[tt]>=inputfuns[0]) {
							sum[tt] += Ahat[jj] * (delta) * ( exp(inputfuns[3+jj]*(times[tt]-inputfuns[0]))-exp(-p[ii+1]*(times[tt]-inputfuns[0])) );
							Jb[tt]  += Ahat[jj] * (delta) * ( exp(inputfuns[3+jj]*(times[tt]-inputfuns[0]))-exp(-p[ii+1]*(times[tt]-inputfuns[0])) );
							Jl[tt]  += Abar[jj] * (delta) * (delta * ( exp(-p[ii+1]*(times[tt]-inputfuns[0]))-exp(inputfuns[3+jj]*(times[tt]-inputfuns[0])))  +  (times[tt]-inputfuns[0]) * exp(-p[ii+1]*(times[tt]-inputfuns[0])));
						}
					}
				}

				for (uint tt=0; tt<${T}; ++tt) {
					if (times[tt]>=inputfuns[0]) {
						TAC[tt] += p[ii] * (sum[tt] + ((inputfuns[1]*(times[tt]-inputfuns[0]))*delta0) *exp(inputfuns[3]*(times[tt]-inputfuns[0])));
						jac[INDEX_JAC_TIME(idx,tt,ii+1)] = (1-aux_par[INDEX_PARAM(idx,0)]) * (Jb[tt] + ((inputfuns[1]*(times[tt]-inputfuns[0]))*delta0) *exp(inputfuns[3]*(times[tt]-inputfuns[0])));
						jac[INDEX_JAC_TIME(idx,tt,ii+2)] = (1-aux_par[INDEX_PARAM(idx,0)]) * (p[ii] * (Jl[tt] + ( exp(-p[ii+1]*(times[tt]-inputfuns[0]))-exp(inputfuns[3]*(times[tt]-inputfuns[0]))) * (inputfuns[1] * (delta0*delta0)) * ((times[tt]-inputfuns[0]) + 2*delta0 )  ));
					}
				}

			}
			//__syncthreads();
			for (uint tt=0; tt<${T}; ++tt) {
				jac[INDEX_JAC_TIME(idx,tt,0)] = IF[tt] - TAC[tt];
				TAC[tt]  = ((1-aux_par[INDEX_PARAM(idx,0)]) * TAC[tt]) + (aux_par[INDEX_PARAM(idx,0)] * IF[tt]);
				if (TAC[tt] < 0.0) {
					TAC[tt] = 1e-16;
				}
				func[INDEX_VOL_TIME(idx,tt)] += TAC[tt];
			}
			//__syncthreads();
		}
	}
}
